#include "hip/hip_runtime.h"
#include <ppl.h>
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include <algorithm>
#include <commonMacro.h>
#include <OpencvCommon.h>
using namespace concurrency;


__global__
void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}

__global__
void filter(int width, int height, int channel, unsigned char *src, unsigned char *dst)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	if (r < height && c < width)
	{
		int colorIdx = (r*width + c) * channel;
		dst[colorIdx] = src[colorIdx];
		dst[colorIdx + 1] = src[colorIdx + 1];
		dst[colorIdx + 2] = src[colorIdx + 2];
	}
}


int main(void)
{
	cv::Mat img = cv::imread("IMG_1772.jpg");
	cv::Mat dstImg(img.size(), img.type());

	unsigned char *d_src, *d_dst;
	int elementN = img.cols * img.rows * img.channels();
	hipMalloc(&d_src, elementN * sizeof(unsigned char));
	hipMalloc(&d_dst, elementN * sizeof(unsigned char));

	hipMemcpy(d_src, img.data, elementN * sizeof(unsigned char), hipMemcpyHostToDevice);
	dim3 dimBlock(32, 32);
	dim3 dimGrid(std::ceil(img.cols / dimBlock.x),
				 std::ceil(img.rows / dimBlock.y));
	HL_INTERVAL_START;
	filter << < dimGrid, dimBlock >> > (img.cols, img.rows, img.channels(), d_src, d_dst);
	HL_INTERVAL_ENDSTR("CUDA IMAGE FILTER");

	HL_INTERVAL_START;
	hipMemcpy(dstImg.data, d_dst, elementN * sizeof(unsigned char), hipMemcpyDeviceToHost);
	HL_INTERVAL_ENDSTR("CUDA IMAGE COPY");

	int w = img.cols, h = img.rows, channel = img.channels();
	int colorW = w * channel;
	HL_INTERVAL_START;
	for (size_t i = 0, r = 0; i < w; i++, r += colorW)
	{
		for (size_t j = 0, c = 0; j < h; j++, c+= 3)
		{
			memcpy(dstImg.data + r + c, img.data + r + c, 3);
		}
	}
	HL_INTERVAL_ENDSTR("CPU IMAGE COPY");

	HL_INTERVAL_START;
	parallel_for(size_t(0), size_t(w), [&](size_t i)
	{
		int r = colorW * i;
		for (size_t j = 0, c = 0; j < h; j++, c += 3)
		{
			memcpy(dstImg.data + r + c, img.data + r + c, 3);
		}
	});
	/*memcpy_s(dstImg.data, elementN * sizeof(unsigned char),
			 img.data, elementN * sizeof(unsigned char));*/
	HL_INTERVAL_ENDSTR("CPU-Paralleled IMAGE COPY");

	/*HL_INTERVAL_START;
	img.copyTo(dstImg);
	HL_INTERVAL_ENDSTR("CV-CPU IMAGE COPY");*/

	HL_INTERVAL_START;
	int numTest = 1 << 20;
	std::vector<float> v1(numTest), v2(numTest);
	for (int i = 0; i < numTest; i++)
	{
		v1[i] = i;
		v2[i] = i;
	}
	HL_INTERVAL_ENDSTR("RELEASE-DEBUG TEST");

	cv::imshow("result", dstImg);
	cv::waitKey(0);

	hipFree(d_src);
	hipFree(d_dst);
	
}